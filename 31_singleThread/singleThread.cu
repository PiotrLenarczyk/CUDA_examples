
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;
typedef uint32_t uint;
#define H2D hipMemcpyHostToDevice 
#define D2H hipMemcpyDeviceToHost
#define OK hipSuccess

//CPU
uint i = 0, ind = 0;
//const uint N = 9 * 1024;
const uint N = 9 * 1024 * 1024;
const uint NBytes_f32 = sizeof( float ) * N;
const uint nArrays = 1;                             //single default stream of 1D array
float *h_arr[ nArrays ], *h_result[ nArrays ];      //pinned H2D && D2H transfers
float nonPinnedArr[ N ];							//non-pinned H2D && D2H transfers are about 2-3times slower via PCIe
float2 *h_f2;
float3 *h_f3;
float4 *h_f4;	              						//float4 host ptr ("http://roxlu.com/2013/011/basic-cuda-example")

//GPU
float *d_arr[ nArrays ];
__device__ float2 d_arr2[ N / 2 ];
__device__ float3 d_arr3[ N / 3 ];
__device__ float4 d_arr4[ N / 4 ];
float2 *d_f2;
float3 *d_f3;
float4 *d_f4;							            //float4 device ptr
const uint nThreads = 512, nBlocks = ( N / nThreads ) + 1;
int freeGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        //HOST
        hipHostFree( h_arr[ i ] );
        hipHostFree( h_result[ i ] );
        hipHostFree( nonPinnedArr );
        hipFree( h_f2 );
        hipFree( h_f3 );
        hipFree( h_f4 );
        //DEVICE
        hipFree( d_arr[ i ] );
        hipFree( d_arr2 );
        hipFree( d_arr3 );
        hipFree( d_arr4 );
        hipFree( d_f2 );
        hipFree( d_f3 );
        hipFree( d_f4 );

    };
    hipDeviceReset();
    return 0;
};

void initGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        if ( hipHostMalloc( ( void** ) &h_arr[ i ], NBytes_f32 , hipHostMallocDefault) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipHostMalloc( ( void** ) &h_result[ i ], NBytes_f32 , hipHostMallocDefault) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipMalloc( ( void** ) &d_arr, NBytes_f32 ) != OK ) { printf( "hipMalloc err!\n" ); return; };
        for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
        auto t1 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
        auto t2 = chrono::high_resolution_clock::now();
        cout << "CPU pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";

        auto t3 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
        auto t4 = chrono::high_resolution_clock::now();
        cout << "CPU non-pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
            << " [ns]\n";
//============================================================================
		for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );
		hipEventRecord( start1 );
		for ( ind = 0; ind < N; ind++ )
		    h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
		hipEventRecord( stop1 );
		hipEventSynchronize( stop1 );
		float milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start1, stop1 );
		cout << "CPU pinned accesses took <cudaEvent> : "
		     << milliseconds * 1000000.0f
		     << " [ns]\n";
		hipEvent_t start2, stop2;
		hipEventCreate( &start2 );
		hipEventCreate( &stop2 );
		hipEventRecord( start2 );
		for ( ind = 0; ind < N; ind++ )
		    nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
		hipEventRecord( stop2 );
		hipEventSynchronize( stop2 );
		milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start2, stop2 );
		cout << "CPU non-pinned accesses took <cudaEvent> : "
		     << milliseconds * 1000000.0f
		     << " [ns]\n";       
//============================================================================              
        h_f2 = ( float2* )malloc( NBytes_f32 );
    	for ( ind = 0; ind < N; ind++ )
    	{
    	if ( ( ind % 2 ) == 0 )
            h_f2[ ind / 2 ].x = h_arr[ i ][ ind ];
        else if ( ( ind % 2 ) == 1 )
            h_f2[ ind / 2 ].y = h_arr[ i ][ ind ];
    	};
    	d_f2 = h_f2;
		if ( hipMalloc( &d_f2, NBytes_f32 ) != OK ) { printf( "hipMalloc err!" ); return; };
    	if ( hipMemcpy( d_f2, h_f2, NBytes_f32, H2D ) != OK ) { printf( "hipMemcpy err!" ); return; };
        h_f3 = ( float3* )malloc( NBytes_f32 );
    	for ( ind = 0; ind < N; ind++ )
    	{
    	if ( ( ind % 3 ) == 0 )
            h_f3[ ind / 3 ].x = h_arr[ i ][ ind ];
        else if ( ( ind % 3 ) == 1 )
            h_f3[ ind / 3 ].y = h_arr[ i ][ ind ];
        else if ( ( ind % 3 ) == 2 )
            h_f3[ ind / 3 ].z = h_arr[ i ][ ind ];
    	};
    	d_f3 = h_f3;
		if ( hipMalloc( &d_f3, NBytes_f32 ) != OK ) { printf( "hipMalloc err!" ); return; };
    	if ( hipMemcpy( d_f3, h_f3, NBytes_f32, H2D ) != OK ) { printf( "hipMemcpy err!" ); return; };
    	h_f4 = ( float4* )malloc( NBytes_f32 );
    	for ( ind = 0; ind < N; ind++ )
    	{
    	if ( ( ind % 4 ) == 0 )
            h_f4[ ind / 4 ].x = h_arr[ i ][ ind ];
        else if ( ( ind % 4 ) == 1 )
            h_f4[ ind / 4 ].y = h_arr[ i ][ ind ];
        else if ( ( ind % 4 ) == 2 )
            h_f4[ ind / 4 ].z = h_arr[ i ][ ind ];
        else if ( ( ind % 4 ) == 3 )
            h_f4[ ind / 4 ].w = h_arr[ i ][ ind ];
    	};
    	d_f4 = h_f4;
		if ( hipMalloc( &d_f4, NBytes_f32 ) != OK ) { printf( "hipMalloc err!" ); return; };
    	if ( hipMemcpy( d_f4, h_f4, NBytes_f32, H2D ) != OK ) { printf( "hipMemcpy err!" ); return; };
		for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] = float( ind );
        for ( ind = 0; ind < 3; ind++ )
            cout << "h_arr[" << ind << "]: " << h_arr[ 0 ][ ind ] << endl;
        hipMemcpy( d_arr[ i ], h_arr[ i ], NBytes_f32, H2D );
    };
};

__global__ void nop( const uint N )
{
    uint a = 0;
    for ( size_t i = 0; i < N; i++ );
        a++;
};

__global__ void singleThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        for ( uint l = 0; l < N; l++ )
            d_in[ l ] += d_in[ l ];
    };
};

__global__ void medianThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        d_in[ tdx ] += d_in[ tdx ];
    };
};

__global__ void makeFloat2( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( !( tdx % 2 ) )
            d_arr2[ tdx / 2 ].x = d_in[ tdx ];
        else
            d_arr2[ tdx / 2 ].y = d_in[ tdx ];
    };
};

__global__ void float2_Access( void )
{
//     for ( uint i = 0; i < 3; i++ )
//         printf( "d_arr2[%i].x: %f\nd_arr2[%i].y: %f\n", i, d_arr2[ i ].x, i, d_arr2[ i ].y );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 2 ) )
    {
        d_arr2[ tdx ].x += d_arr2[ tdx ].x;
        d_arr2[ tdx ].y += d_arr2[ tdx ].y;
    };
};

__global__ void makeFloat3( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 3 ) == 0 )
            d_arr3[ tdx / 3 ].x = d_in[ tdx ];
        else if ( ( tdx % 3 ) == 1 )
            d_arr3[ tdx / 3 ].y = d_in[ tdx ];
        else if ( ( tdx % 3 ) == 2 )
            d_arr3[ tdx / 3 ].z = d_in[ tdx ];
    };
};

__global__ void float3_Access( void )
{
//     for ( uint i = 0; i < 2; i++ )
//         printf( "d_arr3[%i].x: %f\nd_arr3[%i].y: %f\nd_arr3[%i].z: %f\n", i, d_arr3[ i ].x, i, d_arr3[ i ].y, i, d_arr3[ i ].z );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 3 ) )
    {
        d_arr3[ tdx ].x += d_arr3[ tdx ].x;
        d_arr3[ tdx ].y += d_arr3[ tdx ].y;
        d_arr3[ tdx ].z += d_arr3[ tdx ].z;
    };
};

__global__ void makeFloat4( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 4 ) == 0 )
            d_arr4[ tdx / 4 ].x = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 1 )
            d_arr4[ tdx / 4 ].y = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 2 )
            d_arr4[ tdx / 4 ].z = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 3 )
            d_arr4[ tdx / 4 ].w = d_in[ tdx ];
    };
};

__global__ void float4_Access( void )
{
// for ( uint i = 0; i < 2; i++ )
//    printf( "d_inf4[%i].x: %f\nd_inf4[%i].y: %f\nd_inf4[%i].z: %f\nd_inf4[%i].w: %f\n", i, d_inf4[ i ].x, i, d_inf4[ i ].y, i, d_inf4[ i ].z, i, d_inf4[ i ].w );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 4 ) )
    {
	    d_arr4[ tdx ].x += d_arr4[ tdx ].x;
        d_arr4[ tdx ].y += d_arr4[ tdx ].y;
        d_arr4[ tdx ].z += d_arr4[ tdx ].z;
        d_arr4[ tdx ].w += d_arr4[ tdx ].w;
    };
};

__global__ void arrFloat2_Access( float2 *d_in2 )
{
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 2 ) )
    {
	    d_in2[ tdx ].x += d_in2[ tdx ].x;
        d_in2[ tdx ].y += d_in2[ tdx ].y;
    };
};

__global__ void arrFloat3_Access( float3 *d_in3 )
{
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 3 ) )
    {
	    d_in3[ tdx ].x += d_in3[ tdx ].x;
        d_in3[ tdx ].y += d_in3[ tdx ].y;
        d_in3[ tdx ].z += d_in3[ tdx ].z;
    };
};

__global__ void arrFloat4_Access( float4 *d_inf4 )
{
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < ( N / 4 ) )
    {
	    d_inf4[ tdx ].x += d_inf4[ tdx ].x;
        d_inf4[ tdx ].y += d_inf4[ tdx ].y;
        d_inf4[ tdx ].z += d_inf4[ tdx ].z;
        d_inf4[ tdx ].w += d_inf4[ tdx ].w;
    };
};

int main( void )
{
    int gpuCount = 0;
    hipGetDeviceCount( &gpuCount );

    for ( size_t gpuNo = 0; gpuNo < gpuCount; gpuNo++ )
    {
        hipSetDevice( gpuNo );
        hipDeviceProp_t gpuProperties; hipGetDeviceProperties( &gpuProperties, gpuNo ); cout << endl << gpuProperties.name << ": " << endl;
        initGPUMem();
        
        for( i = 0; i < nArrays; i++ )
        {    
            auto f1 = chrono::high_resolution_clock::now();
                makeFloat2<<< nBlocks, nThreads >>>( d_arr[ i ] );
                float2_Access<<< ( N / 2 ) / nThreads, nThreads >>>();
                hipDeviceSynchronize();
            auto f2 = chrono::high_resolution_clock::now();
            cout << "GPU split float2_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f2 - f1 ).count()
                << " [ns]\n";   
            auto f3 = chrono::high_resolution_clock::now();
                makeFloat3<<< nBlocks, nThreads >>>( d_arr[ i ] );
                float3_Access<<< ( N / 3 ) / nThreads, nThreads >>>();        
                hipDeviceSynchronize();
            auto f4 = chrono::high_resolution_clock::now();
            cout << "GPU split float3_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f4 - f3 ).count()
                << " [ns]\n";
            auto f5 = chrono::high_resolution_clock::now();
                makeFloat4<<< nBlocks, nThreads >>>( d_arr[ i ] );     
                float4_Access<<< ( N / 4 ) / nThreads, nThreads >>>();         
                hipDeviceSynchronize();
            auto f6 = chrono::high_resolution_clock::now();
            cout << "GPU split float4_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f6 - f5 ).count()
                << " [ns]\n";   
            auto f7 = chrono::high_resolution_clock::now();
                arrFloat2_Access<<< ( N / 2 ) / nThreads, nThreads >>>( d_f2 );       
                hipDeviceSynchronize();
            auto f8 = chrono::high_resolution_clock::now();
            cout << "GPU pinned arrFloat2_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f8 - f7 ).count()
                << " [ns]\n";
            auto f9 = chrono::high_resolution_clock::now();
                arrFloat3_Access<<< ( N / 3 ) / nThreads, nThreads >>>( d_f3 );       
                hipDeviceSynchronize();
            auto f10 = chrono::high_resolution_clock::now();
            cout << "GPU pinned arrFloat3_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f10 - f9 ).count()
                << " [ns]\n";
            auto f11 = chrono::high_resolution_clock::now();
                arrFloat4_Access<<< ( N / 4 ) / nThreads, nThreads >>>( d_f4 );       
                hipDeviceSynchronize();
            auto f12 = chrono::high_resolution_clock::now();
            cout << "GPU pinned arrFloat4_Access took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( f12 - f11 ).count()
                << " [ns]\n";
            auto t1 = chrono::high_resolution_clock::now();
                singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
                hipDeviceSynchronize();
            auto t2 = chrono::high_resolution_clock::now();
            cout << "single thread GPU accesses took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
                << " [ns]\n";
            auto t3 = chrono::high_resolution_clock::now();
                medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] );
                hipDeviceSynchronize();
            auto t4 = chrono::high_resolution_clock::now();
            cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <chrono> : "
                << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
                << " [ns]\n";
    //============================================================================  
            hipEvent_t start1, stop1;
            hipEventCreate( &start1 );
            hipEventCreate( &stop1 );                  
            hipEventRecord( start1 );
                makeFloat2<<< nBlocks, nThreads >>>( d_arr[ i ] );
                float2_Access<<< ( N / 2 ) / nThreads, nThreads >>>();
                hipDeviceSynchronize();
            hipEventRecord( stop1 );
            hipEventSynchronize( stop1 );
            float milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start1, stop1 );
            cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU split float2_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            hipEvent_t start2, stop2;
            hipEventCreate( &start2 );
            hipEventCreate( &stop2 );                  
            hipEventRecord( start2 );
                makeFloat3<<< nBlocks, nThreads >>>( d_arr[ i ] );
                float3_Access<<< ( N / 3 ) / nThreads, nThreads >>>();
                hipDeviceSynchronize();
            hipEventRecord( stop2 );
            hipEventSynchronize( stop2 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start2, stop2 );
            cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU split float3_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            hipEvent_t start3, stop3;
            hipEventCreate( &start3 );
            hipEventCreate( &stop3 );                  
            hipEventRecord( start3 );
                makeFloat4<<< nBlocks, nThreads >>>( d_arr[ i ] );
                float4_Access<<< ( N / 4 ) / nThreads, nThreads >>>();
                hipDeviceSynchronize();
            hipEventRecord( stop3 );
            hipEventSynchronize( stop3 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start3, stop3 );
            cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU split float4_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            hipEvent_t start4, stop4;
            hipEventCreate( &start4 );
            hipEventCreate( &stop4 );                  
            hipEventRecord( start4 );
                arrFloat2_Access<<< ( N / 2 ) / nThreads, nThreads >>>( d_f2 );   
                hipDeviceSynchronize();
            hipEventRecord( stop4 );
            hipEventSynchronize( stop4 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start4, stop4 );
            cout << "nBlocks[" << ( N / 2 ) / nThreads << "]; nThreads[" << nThreads << "]; GPU pinned arrFloat2_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            hipEvent_t start5, stop5;
            hipEventCreate( &start5 );
            hipEventCreate( &stop5 );                  
            hipEventRecord( start5 );
                arrFloat3_Access<<< ( N / 3 ) / nThreads, nThreads >>>( d_f3 );   
                hipDeviceSynchronize();
            hipEventRecord( stop5 );
            hipEventSynchronize( stop5 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start5, stop5 );
            cout << "nBlocks[" << ( N / 3 ) / nThreads << "]; nThreads[" << nThreads << "]; GPU pinned arrFloat3_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            hipEvent_t start6, stop6;
            hipEventCreate( &start6 );
            hipEventCreate( &stop6 );                  
            hipEventRecord( start6 );
                arrFloat4_Access<<< ( N / 4 ) / nThreads, nThreads >>>(d_f4);
                hipDeviceSynchronize();
            hipEventRecord( stop6 );
            hipEventSynchronize( stop6 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start6, stop6 );
            cout << "nBlocks[" << ( N / 4 ) / nThreads << "]; nThreads[" << nThreads << "]; GPU pinned arrFloat4_Access took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";                
            hipEvent_t start7, stop7;
            hipEventCreate( &start7 );
            hipEventCreate( &stop7 );                  
            hipEventRecord( start7 );
                singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );               
                hipDeviceSynchronize();
            hipEventRecord( stop7 );
            hipEventSynchronize( stop7 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start7, stop7 );
            cout << "single thread GPU accesses took <cudaEvent> : " << milliseconds * 1000000.0f << "[ns]\n";                
            hipEvent_t start8, stop8;
            hipEventCreate( &start8 );
            hipEventCreate( &stop8 );                  
            hipEventRecord( start8 );
                medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] ); 
                hipDeviceSynchronize();
            hipEventRecord( stop8 );
            hipEventSynchronize( stop8 );
            milliseconds = 0.0f;
            hipEventElapsedTime( &milliseconds, start8, stop8 );
            cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <cudaEvent> : " 
                << milliseconds * 1000000.0f << "[ns]\n";
            nop<<< nBlocks, nThreads >>>( N );
    //============================================================================                
            hipMemcpy( h_result[ i ], d_arr[ i ], NBytes_f32, D2H );//cudaMemcpyFromSymbol( h_result[ i ], d_arr2, NBytes_f32, H2D );
        };
        for ( ind = 0; ind < 3; ind++ )
            cout << "   h_result[" << ind << "]: " << h_result[ 0 ][ ind ] << endl;
        
        freeGPUMem();
    }; //end of gpuCount
    
    return 0;
}

