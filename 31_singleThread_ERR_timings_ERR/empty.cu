#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

using namespace std;
typedef uint32_t uint;
#define H2D hipMemcpyHostToDevice 
#define D2H hipMemcpyDeviceToHost
#define OK hipSuccess

//CPU
uint i = 0, ind = 0;
const uint N = 9 * 1024 * 1024;
const uint NBytes_f32 = sizeof( float ) * N;
const uint nArrays = 1;                             //single default stream of 1D array
float *h_arr[ nArrays ], *h_result[ nArrays ];      //pinned H2D && D2H transfers
float nonPinnedArr[ N ];							//non-pinned H2D && D2H transfers are about 2-3times slower via PCIe

//GPU
float *d_arr[ nArrays ];
__device__ float2 d_arr2[ N / 2 ];
__device__ float3 d_arr3[ N / 3 ];
__device__ float4 d_arr4[ N / 4 ];
const uint nThreads = 512, nBlocks = ( N / nThreads ) + 1;
int freeGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        //HOST
        hipHostFree( h_arr[ i ] );
        hipHostFree( h_result[ i ] );
        hipHostFree( nonPinnedArr );
        //DEVICE
        hipFree( d_arr[ i ] );
        hipFree( d_arr2 );
        hipFree( d_arr3 );
        hipFree( d_arr4 );
    };
    hipDeviceReset();
    return 0;
};

void initGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        if ( hipHostMalloc( ( void** ) &h_arr[ i ], NBytes_f32 ) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipHostMalloc( ( void** ) &h_result[ i ], NBytes_f32 ) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipMalloc( ( void** ) &d_arr, NBytes_f32 ) != OK ) { printf( "hipMalloc err!\n" ); return; };
        for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
        auto t1 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
        auto t2 = chrono::high_resolution_clock::now();
        cout << "CPU pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";

        auto t3 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
        auto t4 = chrono::high_resolution_clock::now();
        cout << "CPU non-pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
            << " [ns]\n";
//============================================================================
		for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );
		hipEventRecord( start1 );
		for ( ind = 0; ind < N; ind++ )
		    h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
		hipEventRecord( stop1 );
		hipEventSynchronize( stop1 );
		float milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start1, stop1 );
		cout << "CPU pinned accesses took <cudaEvent> : "
		     << milliseconds * 1000000.0f
		     << " [ns]\n";
		hipEvent_t start2, stop2;
		hipEventCreate( &start2 );
		hipEventCreate( &stop2 );
		hipEventRecord( start2 );
		for ( ind = 0; ind < N; ind++ )
		    nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
		hipEventRecord( stop2 );
		hipEventSynchronize( stop2 );
		milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start2, stop2 );
		cout << "CPU non-pinned accesses took <cudaEvent> : "
		     << milliseconds * 1000000.0f
		     << " [ns]\n";       
//============================================================================        
//CPU/GPU speedups of memory accesses:
//	-worst case scenario of memory utilization: 1th CPU <-> 1th GPU {<chrono>, <cudaEvent>}
//	-typical scenario 1th CPU <-> parallel th GPU {<chrono>, <cudaEvent>}
//	-CPU memory accesses via float4 - check
//	-GDDR5 GPU memory accesses opimalization with tuples of sizes = {1,2,3,4}
//	-both GPU's hipSetDevice();
//makeFloat2 vs float[ N/2 ][ 2 ]
//      ...                  
		for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] = float( ind );
        for ( ind = 0; ind < 3; ind++ )
            cout << "h_arr[" << ind << "]: " << h_arr[ 0 ][ ind ] << endl;
        hipMemcpy( d_arr[ i ], h_arr[ i ], NBytes_f32, H2D );
    };
};

__global__ void singleThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        for ( uint l = 0; l < N; l++ )
            d_in[ l ] += d_in[ l ];
    };
};

__global__ void medianThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        d_in[ tdx ] += d_in[ tdx ];
    };
};

__global__ void makeFloat2( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( !( tdx % 2 ) )
            d_arr2[ tdx / 2 ].x = d_in[ tdx ];
        else
            d_arr2[ tdx / 2 ].y = d_in[ tdx ];
    };
};

__global__ void float2_Access( void )
{
//     for ( uint i = 0; i < 3; i++ )
//         printf( "d_arr2[%i].x: %f\nd_arr2[%i].y: %f\n", i, d_arr2[ i ].x, i, d_arr2[ i ].y );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( !( tdx % 2 ) )
            d_arr2[ tdx / 2 ].x += d_arr2[ tdx / 2 ].x;
        else
            d_arr2[ tdx / 2 ].y = d_arr2[ tdx / 2 ].y;
    };
};

__global__ void makeFloat3( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 3 ) == 0 )
            d_arr3[ tdx / 3 ].x = d_in[ tdx ];
        else if ( ( tdx % 3 ) == 1 )
            d_arr3[ tdx / 3 ].y = d_in[ tdx ];
        else if ( ( tdx % 3 ) == 2 )
            d_arr3[ tdx / 3 ].z = d_in[ tdx ];
    };
};

__global__ void float3_Access( void )
{
//     for ( uint i = 0; i < 2; i++ )
//         printf( "d_arr3[%i].x: %f\nd_arr3[%i].y: %f\nd_arr3[%i].z: %f\n", i, d_arr3[ i ].x, i, d_arr3[ i ].y, i, d_arr3[ i ].z );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 3 ) == 0 )
            d_arr3[ tdx / 3 ].x += d_arr3[ tdx / 3 ].x;
        else if ( ( tdx % 3 ) == 1 )
            d_arr3[ tdx / 3 ].y += d_arr3[ tdx / 3 ].y;
        else if ( ( tdx % 3 ) == 2 )
            d_arr3[ tdx / 3 ].z += d_arr3[ tdx / 3 ].z;
    };
};

__global__ void makeFloat4( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 4 ) == 0 )
            d_arr4[ tdx / 4 ].x = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 1 )
            d_arr4[ tdx / 4 ].y = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 2 )
            d_arr4[ tdx / 4 ].z = d_in[ tdx ];
        else if ( ( tdx % 4 ) == 3 )
            d_arr4[ tdx / 4 ].w = d_in[ tdx ];
    };
};

__global__ void float4_Access( void )
{
//     for ( uint i = 0; i < 2; i++ )
//         printf( "d_arr4[%i].x: %f\nd_arr4[%i].y: %f\nd_arr4[%i].z: %f\nd_arr4[%i].w: %f\n", i, d_arr4[ i ].x, i, d_arr4[ i ].y, i, d_arr4[ i ].z, i, d_arr4[ i ].w );
    uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        if ( ( tdx % 4 ) == 0 )
            d_arr4[ tdx / 4 ].x += d_arr4[ tdx / 4 ].x;
        else if ( ( tdx % 4 ) == 1 )
            d_arr4[ tdx / 4 ].y += d_arr4[ tdx / 4 ].y;
        else if ( ( tdx % 4 ) == 2 )
            d_arr4[ tdx / 4 ].z += d_arr4[ tdx / 4 ].z;
        else if ( ( tdx % 4 ) == 3 )
            d_arr4[ tdx / 4 ].w += d_arr4[ tdx / 4 ].w;
    };
};

int main( void )
{
    initGPUMem();
    
    for( i = 0; i < nArrays; i++ )
    {
    
    /*
		http://roxlu.com/2013/011/basic-cuda-example
	*/
    	float4 *h_f4, *d_f4;
    	*h_f4 = ( float4* )malloc( NBytes_f32 );
    	for ( ind = 0; ind < N; ind++ )
    	{
    	if ( ( ind % 4 ) == 0 )
            h_f4[ ind / 4 ].x = h_arr[ ind ];
        else if ( ( ind % 4 ) == 1 )
            h_f4[ ind / 4 ].y = h_arr[ ind ];
        else if ( ( ind % 4 ) == 2 )
            h_f4[ ind / 4 ].z = h_arr[ ind ];
        else if ( ( ind % 4 ) == 3 )
            h_f4[ ind / 4 ].w = h_arr[ ind ];
    	};
    	d_f4 = f_f4;
		if ( hipMalloc( d_f4, NBytes_f32 ) != OK ) { printf( "hipMalloc err!" ); return; };
    	if ( hipMemcpy( d_f4, h_f4, NBytes_f32 ) != OK ) { printf( "hipMemcpy err!" ); return; };
    
    
        auto f1 = chrono::high_resolution_clock::now();
            makeFloat2<<< nBlocks, nThreads >>>( d_arr[ i ] );
            float2_Access<<< nBlocks, nThreads >>>();
            hipDeviceSynchronize();
        auto f2 = chrono::high_resolution_clock::now();
        cout << "GPU float2_Access took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( f2 - f1 ).count()
            << " [ns]\n";   
        auto f3 = chrono::high_resolution_clock::now();
            makeFloat3<<< nBlocks, nThreads >>>( d_arr[ i ] );
            float3_Access<<< nBlocks, nThreads >>>();        
            hipDeviceSynchronize();
        auto f4 = chrono::high_resolution_clock::now();
        cout << "GPU float3_Access took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( f4 - f3 ).count()
            << " [ns]\n";
        auto f5 = chrono::high_resolution_clock::now();
            makeFloat4<<< nBlocks, nThreads >>>( d_arr[ i ] );
            float4_Access<<< nBlocks, nThreads >>>();         
            hipDeviceSynchronize();
        auto f6 = chrono::high_resolution_clock::now();
        cout << "GPU float4_Access took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( f6 - f5 ).count()
            << " [ns]\n";    
        auto t1 = chrono::high_resolution_clock::now();
            singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
            hipDeviceSynchronize();
        auto t2 = chrono::high_resolution_clock::now();
        cout << "single thread GPU accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";
        auto t3 = chrono::high_resolution_clock::now();
            medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] );
            hipDeviceSynchronize();
        auto t4 = chrono::high_resolution_clock::now();
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <chrono> : "
             << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
             << " [ns]\n";
//============================================================================  
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );                  
        hipEventRecord( start1 );
        	singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
            hipDeviceSynchronize();
        hipEventRecord( stop1 );
        hipEventSynchronize( stop1 );
        float milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start1, stop1 );
        cout << "single thread GPU accesses took <cudaEvent> : " << milliseconds * 1000000.0f << "[ns]\n";
		hipEvent_t start2, stop2;
		hipEventCreate( &start2 );
		hipEventCreate( &stop2 );                  
        hipEventRecord( start2 );
        	medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] );
            hipDeviceSynchronize();
        hipEventRecord( stop2 );
        hipEventSynchronize( stop2 );
        milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start2, stop2 );
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <cudaEvent> : " 
        	 << milliseconds * 1000000.0f << "[ns]\n";
             
//============================================================================                
        hipMemcpy( h_result[ i ], d_arr[ i ], NBytes_f32, D2H );
    hipMemcpyFromSymbol( h_result[ i ], HIP_SYMBOL(d_arr2), NBytes_f32, H2D );
    };
    for ( ind = 0; ind < 3; ind++ )
        cout << "   h_result[" << ind << "]: " << h_result[ 0 ][ ind ] << endl;
    
	return freeGPUMem();
}
