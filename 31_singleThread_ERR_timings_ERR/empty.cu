#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

using namespace std;
typedef uint32_t uint;
#define H2D hipMemcpyHostToDevice 
#define D2H hipMemcpyDeviceToHost
#define OK hipSuccess

//CPU
uint i = 0, ind = 0;
const uint N = 9 * 1024 * 1024;
const uint NBytes_f32 = sizeof( float ) * N;
const uint nArrays = 1;                             //single default stream of 1D array
float *h_arr[ nArrays ], *h_result[ nArrays ];      //pinned H2D && D2H transfers
float nonPinnedArr[ N ];							//non-pinned H2D && D2H transfers are about 2-3times slower via PCIe

//GPU
float *d_arr[ nArrays ];
__device__ float2 d_arr2[ N / 2 ];
__device__ float3 d_arr3[ N / 3 ];
__device__ float4 d_arr4[ N / 4 ];
const uint nThreads = 512, nBlocks = ( N / nThreads ) + 1;
int freeGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        //HOST
        hipHostFree( h_arr[ i ] );
        hipHostFree( h_result[ i ] );
        hipHostFree( nonPinnedArr );
        //DEVICE
        hipFree( d_arr[ i ] );
        hipFree( d_arr2 );
        hipFree( d_arr3 );
        hipFree( d_arr4 );
    };
    hipDeviceReset();
    return 0;
};

void initGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        if ( hipHostMalloc( ( void** ) &h_arr[ i ], NBytes_f32 ) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipHostMalloc( ( void** ) &h_result[ i ], NBytes_f32 ) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipMalloc( ( void** ) &d_arr, NBytes_f32 ) != OK ) { printf( "hipMalloc err!\n" ); return; };
//      ...
		for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
        auto t1 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
        auto t2 = chrono::high_resolution_clock::now();
        cout << "CPU pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";
        for ( ind = 0; ind < 3; ind++ )
            cout << "h_arr[" << ind << "]: " << h_arr[ 0 ][ ind ] << endl;
        auto t3 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
        auto t4 = chrono::high_resolution_clock::now();
        cout << "CPU non-pinned accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
            << " [ns]\n";
//============================================================================
		for ( ind = 0; ind < N; ind++ )
		{
            h_arr[ i ][ ind ] = float( ind );
            nonPinnedArr[ ind ] = float( ind );
        };
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );
		hipEventRecord( start1 );
		for ( ind = 0; ind < N; ind++ )
		    h_arr[ i ][ ind ] += h_arr[ i ][ ind ];
		hipEventRecord( stop1 );
		hipEventSynchronize( stop1 );
		float milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start1, stop1 );
		cout << "CPU pinned accesses took <cudaEvent> : "
		     << milliseconds / float( 1E6 )
		     << " [ns]\n";
		hipEvent_t start2, stop2;
		hipEventCreate( &start2 );
		hipEventCreate( &stop2 );
		hipEventRecord( start2 );
		for ( ind = 0; ind < N; ind++ )
		    nonPinnedArr[ ind ] += nonPinnedArr[ ind ];
		hipEventRecord( stop2 );
		hipEventSynchronize( stop2 );
		float milliseconds = 0.0f;
		hipEventElapsedTime( &milliseconds, start2, stop2 );
		cout << "CPU non-pinned accesses took <cudaEvent> : "
		     << milliseconds / float( 1E6 )
		     << " [ns]\n";         
//============================================================================        
//CPU/GPU speedups of memory accesses:
//	-worst case scenario of memory utilization: 1th CPU <-> 1th GPU {<chrono>, <cudaEvent>}
//	-typical scenario 1th CPU <-> parallel th GPU {<chrono>, <cudaEvent>}
//	-CPU memory accesses via float4 - check
//	-GDDR5 GPU memory accesses opimalization with tuples of sizes = {1,2,3,4}
//	-both GPU's hipSetDevice();
//      ...        
		for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] = float( ind );
        hipMemcpy( d_arr[ i ], h_arr[ i ], NBytes_f32, H2D );
    };
};

__global__ void singleThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        for ( uint l = 0; l < N; l++ )
            d_in[ l ] += d_in[ l ];
    };
};

__global__ void medianThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        d_in[ tdx ] += d_in[ tdx ];
    };
};

__global__ void singleTupleAccess

int main( void )
{
    initGPUMem();
    
    for( i = 0; i < nArrays; i++ )
    {
        auto t1 = chrono::high_resolution_clock::now();
        singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
        hipDeviceSynchronize();
        auto t2 = chrono::high_resolution_clock::now();
        cout << "single thread GPU accesses took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";
        auto t3 = chrono::high_resolution_clock::now();
		medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] );
        hipDeviceSynchronize();
        auto t4 = chrono::high_resolution_clock::now();
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <chrono> : "
             << chrono::duration_cast< chrono::nanoseconds >( t4 - t3 ).count()
             << " [ns]\n";
//============================================================================  
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );                  
        hipEventRecord( start1 );
        	singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
        hipEventRecord( stop1 );
        hipEventSynchronize( stop1 );
        float milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start1, stop1 );
        cout << "single thread GPU accesses took <cudaEvent> : " << milliseconds / float( 1E6 ) << "[ns]\n";
		hipEvent_t start2, stop2;
		hipEventCreate( &start2 );
		hipEventCreate( &stop2 );                  
        hipEventRecord( start2 );
        	medianThreadAccess<<< nBlocks, nThreads >>>( d_arr[ i ] );
        hipEventRecord( stop2 );
        hipEventSynchronize( stop2 );
        milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start2, stop2 );
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took <cudaEvent> : " 
        	 << milliseconds / float( 1E6 ) << "[ns]\n";
//============================================================================                
        hipMemcpy( h_result[ i ], d_arr[ i ], NBytes_f32, D2H );
    };
    for ( ind = 0; ind < 3; ind++ )
        cout << "   h_result[" << ind << "]: " << h_result[ 0 ][ ind ] << endl;
    
	return freeGPUMem();
}
