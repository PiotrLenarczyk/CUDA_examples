
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;
typedef uint32_t uint;
#define H2D hipMemcpyHostToDevice 
#define D2H hipMemcpyDeviceToHost
#define OK hipSuccess

//CPU
uint i = 0, ind = 0;
const uint N = 8E6;
const uint NBytes_f32 = sizeof( float ) * N;
const uint nArrays = 1;                             //single default stream of 1D array
float *h_arr[ nArrays ], *h_result[ nArrays ];      //pinned H2D && D2H transfers

//GPU
float *d_arr[ nArrays ];
const uint nThreads = 512, nBlocks = ( N / nThreads ) + 1;
void initGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        if ( hipHostMalloc( ( void** ) &h_arr[ i ], NBytes_f32 , hipHostMallocDefault) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipHostMalloc( ( void** ) &h_result[ i ], NBytes_f32 , hipHostMallocDefault) != OK ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipMalloc( ( void** ) &d_arr, NBytes_f32 ) != OK ) { printf( "hipMalloc err!\n" ); return; };
//      ...
        auto t1 = chrono::high_resolution_clock::now();
        for ( ind = 0; ind < N; ind++ )
            h_arr[ i ][ ind ] = float( ind );
        auto t2 = chrono::high_resolution_clock::now();
        cout << "CPU accesses took "
            << chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count()
            << " [ns]\n";
        for ( ind = 0; ind < 3; ind++ )
            cout << "h_arr[" << ind << "]: " << h_arr[ 0 ][ ind ] << endl;
//CPU/GPU speedups of memory accesses:
//worst case scenario of memory utilization: 1th CPU <-> 1th GPU
//typical scenario 1th CPU <-> parallel th GPU ( with tuples of sizes = {1,2,3,4} )
//both GPU's
//      ...        
        hipMemcpy( d_arr[ i ], h_arr[ i ], NBytes_f32, H2D );
    };
};


int freeGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        //HOST
        hipHostFree( h_arr[ i ] );
        hipHostFree( h_result[ i ] );
        //DEVICE
        hipFree( d_arr[ i ] );
    };
    hipDeviceReset();
    return 0;
};

__global__ void singleThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        for ( uint l = 0; l < N; l++ )
            d_in[ l ] = -( float )l;
    };
};

__global__ void medianThreadAccess( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tdx < N )
    {
        d_in[ tdx ] = -( float )d_in[ tdx ];
    };
};


int main( void )
{
    initGPUMem();
    
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    
    for( i = 0; i < nArrays; i++ )
    {
        hipEventRecord( start );
        singleThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
        hipEventRecord( stop );
        hipEventSynchronize(stop);
        float milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start, stop );
        cout << "single thread GPU accesses took " << milliseconds / 1000.0f << "[ns]\n";
        hipEventRecord( start );
        medianThreadAccess<<< 1, 1 >>>( d_arr[ i ] );
        hipEventRecord( stop );
        hipEventSynchronize(stop);
         milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start, stop );
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU accesses took " << milliseconds / 1000.0f << "[ns]\n";
        hipMemcpy( h_result[ i ], d_arr[ i ], NBytes_f32, D2H );
    };
    for ( ind = 0; ind < 3; ind++ )
        cout << "   h_result[" << ind << "]: " << h_result[ 0 ][ ind ] << endl;
    
	return freeGPUMem();
}
