
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;

//CPU
typedef unsigned int uint;
uint i = 0;
int gpuCount = 0;
void initalizeHost( float *ip, uint size );

//GPU
hipDeviceProp_t gpuProperties;
const uint N = 1E7;
const uint nThreads = 512;
const uint nBlocks = ( N / nThreads ) + 1;
const uint UNROLLING = 16;                   //check [ 8 16 32 64 ]; I would guess sixteen times unrolling;
__global__ void nop(){}; 
#define hipMemcpyHostToDevice H2D;          //useful alias
#define hipMemcpyDeviceToHost D2H;
__global__ void trivialLoop()
{
    uint a = 0;
    for ( uint32_t i = 0; i < N; i++ )
        a = i;
}

__global__ void unrollTrivialLoop()
{
    uint a = 0;
    #pragma unroll UNROLLING                //briliant feature
    for ( uint32_t i = 0; i < N; i++ )
        a = i;
};

int main( void )
{
    hipGetDeviceCount( &gpuCount );
    //HOST
    float *h_arr[ gpuCount ];           //     float **h_arr = ( float** )malloc( sizeof( float * ) * gpuCount ); //alternatively
    uint perDevN = 1E3 / gpuCount;
    //DEVICE
    hipStream_t stream[ gpuCount ];
    float *d_arr[ gpuCount ];           //     float **d_arr = ( float** )malloc( sizeof( float * ) * gpuCount ); //alternatively
    
    //alocate & initialize H,D memories
    for ( i = 0; i < gpuCount; i++ )
    {
        //HOST 
        hipHostMalloc( ( void** ) &h_arr[ i ], perDevN , hipHostMallocDefault);
        initalizeHost( h_arr[ i ], perDevN );
        //DEVICE
        hipSetDevice( i );
        hipMalloc( ( void** ) &d_arr[ i ], perDevN );
        hipStreamCreate( &stream[ i ] );
    }
    
    //DEVICE computations
    for ( i = 0; i < gpuCount; i++ )
    {
        hipSetDevice( i );
        hipGetDeviceProperties( &gpuProperties, i );
        cout << endl << gpuProperties.name << ": " << endl;
        auto t1 = chrono::high_resolution_clock::now();
        trivialLoop<<< 1, 1 >>>();
        nop<<< 1, 1 >>>();
        auto t2 = chrono::high_resolution_clock::now();
        uint elapsed = uint( chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count() ); 
        printf( "loop elapsed: %d \n", elapsed );
        
        t1 = chrono::high_resolution_clock::now();
        unrollTrivialLoop<<< 1, 1 >>>();
        nop<<< 1, 1 >>>();
        t2 = chrono::high_resolution_clock::now();
        elapsed = chrono::duration_cast< chrono::nanoseconds >( t2 - t1 ).count();  
        printf( "unrolled loop elapsed: %d \n", elapsed );
        
        
        
        
        
        //cudaMemcpyAsync( d_arr[i], h_arr[i], H2D, stream[i] );
        //kernel<<< nT, nB, stream[i] >>>( d_arr, perDevN );
        /*    uint tid = threadIdx.x;
              uint idx = blockIdx.x * blockDim.x + threadIdx.x;
              if( idx < perDevN ) :...
        */
        //cudaMemcpyAsync( gpuRef[i], d_arr, D2H stream[i] );
        
        
        
        
        
        
    }
    
    //free memories
    for ( i = 0; i < gpuCount; i++ )
    {
        //HOST 
        hipHostFree( h_arr[ i ] );
        //DEVICE
        hipSetDevice( i );
        hipFree( d_arr[ i ] );
        hipStreamDestroy( stream[ i ] );
    }

    hipDeviceReset();
    return 0;
}

void initalizeHost( float *ip, uint size )
{
    for ( size_t i = 0; i < size; i++ )
        ip[ i ] = 0.0f;
};

//Post Scriptum: In my professional opinion, coprocessors: GTX1080ti is brand-new and off-the-shell optimal; GTX770 is used optimal - I've heard about R9Nano and HD5770 ( GFLOPS/USD; GFLOPS/W; QualityWithBandwidthAndMemSize/Price; ); 
