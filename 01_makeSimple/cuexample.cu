
#include <hip/hip_runtime.h>
__global__ void exampleDevice( float * d )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d[ idx ] = idx;   
}

extern "C" void exampleHost( float * h, int blockDim, int threadDim )
{
    float * d;
    hipMalloc( ( void** )&d, blockDim * threadDim * sizeof( float ) );
    exampleDevice<<<blockDim, threadDim>>>( d );
    hipMemcpy( h, d, blockDim * threadDim * sizeof( float ), hipMemcpyDeviceToHost ); 
}
