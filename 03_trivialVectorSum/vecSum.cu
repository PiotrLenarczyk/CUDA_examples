
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#define N 60000

using namespace std;

__global__ void add( float *a, float *b, float *c )
{
	int tid = blockIdx.x;
	if ( tid < N )
		c[ tid ] = a[ tid ] + b[ tid ];
}

int main ( void )
{
    //host vectors
    vector < float > firstVec( N, 1.11f );
    vector < float > secondVec( N, 3.01f );
    vector < float > resultsVec( N, 0.0f );

	//GPU memory allocation
    float *dev_a, *dev_b, *dev_c;
	hipMalloc( ( void** )&dev_a, N * sizeof( float ) );
	hipMalloc( ( void** )&dev_b, N * sizeof( float ) );
	hipMalloc( ( void** )&dev_c, N * sizeof( float ) );
	
	//copy / download data in direction HostToDevice
	hipMemcpy( dev_a, &firstVec[0], N * sizeof( float ), hipMemcpyHostToDevice );
	hipMemcpy( dev_b, &secondVec[0], N * sizeof( float ), hipMemcpyHostToDevice );
    
	//calculate vectors sum, using Blocks
	add<<<N,1>>> ( dev_a, dev_b, dev_c );
	
	//copy / upload results data c[] in direction DeviceToHost
	hipMemcpy( &resultsVec[0], dev_c, N * sizeof( float ), hipMemcpyDeviceToHost );

	//show results
    for ( int i = 0; i < 5; i++ ) 
        cout << firstVec[ i ] << " + " << secondVec[ i ] << " = " << resultsVec[ i ] << endl;

	//free GPU memory
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
	
    hipDeviceReset();
	return 0;
}
