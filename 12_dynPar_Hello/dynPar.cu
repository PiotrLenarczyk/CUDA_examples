#include "hip/hip_runtime.h"
//STL
#include <stdio.h>

__global__ void childKernel()
{
    printf( "Hello %d", threadIdx.x );
}

__global__ void parentKernel()
{
    childKernel<<< 1, 2 >>>();
    hipDeviceSynchronize();
    printf( "World!\n" );
}

int main( int argc, char *argv[] )
{
    parentKernel<<< 1, 2 >>>();
    hipDeviceSynchronize();
    return 0;
}
