
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define H2D hipMemcpyHostToDevice 
#define D2H hipMemcpyDeviceToHost
#define OK hipSuccess

using namespace std;
typedef uint32_t uint;

//CPU
uint i = 0, ind = 0;
const uint N = 8E3;
const uint NBytes_f32 = sizeof( float ) * N;
const uint nArrays = 1;                             //single default stream of 1D array
float *h_arr[ nArrays ], *h_result[ nArrays ];      //pinned H2D && D2H transfers

//GPU
float *d_arr[ nArrays ];
__device__ float4 d_sArr[ 1 ];	//d_s[].x;.y;.z;.w; cudaMemcpyToSymbol(*dest,*src,byteSize);cudaMemcpyFromSymbol(*dest,*src,byteSize);
const uint nThreads = 512, nBlocks = ( N / nThreads ) + 1;
inline int freeGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        //HOST
        hipHostFree( h_arr[ i ] );
        hipHostFree( h_result[ i ] );
        //DEVICE
        hipFree( d_arr[ i ] );
    };
    hipDeviceReset();
    return 0;
};

inline void initGPUMem( void )
{
    for ( i= 0; i < nArrays; i++ )
    {
        if ( hipHostMalloc( ( void** ) &h_arr[ i ], NBytes_f32 , hipHostMallocDefault) != hipSuccess ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipHostMalloc( ( void** ) &h_result[ i ], NBytes_f32 , hipHostMallocDefault) != hipSuccess ) { printf( "hipHostMalloc err!\n" ); return; };
        if ( hipMalloc( ( void** ) &d_arr[ i ], NBytes_f32 ) != hipSuccess ) { printf( "hipMalloc err!\n" ); return; };
//      ...
//      h_arr[] data load
            for ( ind = 0; ind < N; ind++ )
                h_arr[ i ][ ind ] = float( ind );
            for ( ind = 0; ind < 3; ind++ )
                cout << "h_arr[" << i << "][" << ind << "]: " << h_arr[ i ][ ind ] << endl;
//      ...        
        hipMemcpyAsync( d_arr[ i ], h_arr[ i ], NBytes_f32, H2D );
    };
};

__global__ void emptyKernel( float *d_in )
{
	uint tdx = threadIdx.x + blockIdx.x * blockDim.x; 
    if ( tdx < N )
    {
        printf( "thread[%i].block[%i]\n", tdx, blockDim.x );
    };
};

int main( void )
{
    initGPUMem();
    
    for( i = 0; i < nArrays; i++ )
    {
		auto f1 = chrono::high_resolution_clock::now();
        	emptyKernel<<< 1, 1 >>>( d_arr[ i ] );
			hipDeviceSynchronize();
        auto f2 = chrono::high_resolution_clock::now();
        cout << "GPU kernel took <chrono> : "
            << chrono::duration_cast< chrono::nanoseconds >( f2 - f1 ).count()
            << " [ns]\n"; 
		hipEvent_t start1, stop1;
		hipEventCreate( &start1 );
		hipEventCreate( &stop1 );                  
        hipEventRecord( start1 );
            emptyKernel<<< 1, 1 >>>( d_arr[ i ] );  
            hipDeviceSynchronize();
        hipEventRecord( stop1 );
        hipEventSynchronize( stop1 );
        float milliseconds = 0.0f;
        hipEventElapsedTime( &milliseconds, start1, stop1 );
        cout << "nBlocks[" << nBlocks << "]; nThreads[" << nThreads << "]; GPU kernel took <cudaEvent> : " 
        	 << milliseconds * 1000000.0f << "[ns]\n";
        hipMemcpy( h_result[ i ], d_arr[ i ], NBytes_f32, D2H );
            for ( ind = 0; ind < 3; ind++ )
                cout << "h_result[" << i << "][" << ind << "]: " << h_result[ i ][ ind ] << endl;
    };
    
	return freeGPUMem();
}

